  
#include <hip/hip_runtime.h>
#include <iostream>
  #include <chrono>
 
void handleCudaError(hipError_t cudaERR){ //error handling
  if (cudaERR!=hipSuccess){
    printf("CUDA ERROR : %s\n", hipGetErrorString(cudaERR));
  }
}

  __global__ void polynomial_expansion (float* poly, int degree,
  int n, float* array) {
  	int index = blockIdx.x * blockDim.x + threadIdx.x;
	if( index < n )
  	{
		float out = 0.0;
  		float xtothepowerof = 1.0;
  		for ( int x = 0; x <= degree; ++x)
  		{
  			out += xtothepowerof * poly[x];
  			xtothepowerof *= array[index];
  		}
  		array[index] = out;
  	}
  }

  int main (int argc, char* argv[]) 
  {
  	if (argc < 3) 
  	{
  		std::cerr<<"usage: "<<argv[0]<<" n degree"<<std::endl;
  		return -1;
  	}

	int n = atoi(argv[1]); 
	int degree = atoi(argv[2]);
	int nbiter = 1;

  	float* array = new float[n];
  	float* poly = new float[degree+1];
  	for (int i=0; i<n; ++i){
  		array[i] = 1.;
	}

  	for (int i=0; i<degree+1; ++i){
  		poly[i] = 1.;
	}

  	float *d_array, *d_poly;
  	std::chrono::time_point<std::chrono::system_clock> begin, end;
  	begin = std::chrono::system_clock::now();

  	handleCudaError(hipMalloc(&d_array, n*sizeof(float)));
  	handleCudaError(hipMalloc(&d_poly, (degree+1)*sizeof(float)));

  	hipMemcpy(d_array, array, n*sizeof(float), hipMemcpyHostToDevice); //memory allocation
  	hipMemcpy(d_poly, poly,(degree+1)*sizeof(float), hipMemcpyHostToDevice);

  	polynomial_expansion<<<(n+255)/256, 256>>>(d_poly, degree, n, d_array);
  	hipMemcpy(array, d_array, n*sizeof(float), hipMemcpyDeviceToHost);

  
  	hipFree(d_array);
  	hipFree(d_poly);

  	hipDeviceSynchronize();

	{
	    bool correct = true;
	    int ind;
		    for (int i=0; i< n; ++i) {
			    if (fabs(array[i]-(degree+1))>0.01) {
				    correct = false;
				    ind = i;
			    }
		    }
	    if (!correct)
	    std::cerr<<"Result is incorrect. In particular array["<<ind<<"] should be "<<degree+1<<" not "<< array[ind]<<std::endl;
	}

  	end = std::chrono::system_clock::now();
  	std::chrono::duration<double> totaltime = (end-begin)/nbiter;

  	std::cout<<n<<" "<<degree<<" "<<totaltime.count()<<std::endl;

  	delete[] array;
  	delete[] poly;

  	return 0;
  }
